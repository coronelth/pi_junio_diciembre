
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void llenarVelocidad(double ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvel=9;
	float leer;
	
 	fichero = fopen("matriz_con_func_dist.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvel;j++){

			fscanf(fichero,"%f",&leer);	
			pmat[i][j]=leer;
			printf("%f",leer);
		}
	fscanf(fichero, "\n"); 
	}

	fclose(fichero);
}
