#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



__global__  void sumarvelocidad(double ** pdist,int ** pvec,double ** psum, int node) {
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion
int k;
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
// int offset = x + y * blockDim.x * gridDim.x;

if (x<node){ //para que se paralelice en cada nodo
	if (y<nvec){	//para que se paralelice en cada vecino
		for(k=0;k<ndist;k++){	//para cada velocidad realizo la suma al no saber como paralelizar esta parte
			psum[x][k]+=pdist[pvec[x][y]][k];				
						}
		}


		}

}
