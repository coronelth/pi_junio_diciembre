// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar "velocidad"

void llenarVelocidad(float ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvel=9;
	float leer;

	//float **pmat = NULL;	

 	fichero = fopen("matriz_con_func_dist.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }


	printf("\n\nEstoy por leer del archivo matriz_con_func_dist.txt \n\n");		
	
	for(i=0;i<node;i++){
		
		printf("Estoy por entrar al for con i...");

		for(j=0;j<nvel;j++){

			fscanf(fichero,"%f",&leer);
			printf("Imprimo leer= %f  y cargo pmat[i][j]");	
			pmat[i][j]=leer;
			printf("%f",leer);
		}
	fscanf(fichero, "\n"); 
	}

	fclose(fichero);

	for(i=0;i<node;i++){
		for(j=0;j<nvel;j++){
			printf("%f\t",pmat[i][j]);
		}
	}
}


//-------------Funcion llenar los vecinos

void llenarVecinos(int ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvec=9;
	int aux=0;

	
	/*pmat[k]=0;*/

 	fichero = fopen("matriz_con_vecinos.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvec;j++){

			fscanf(fichero,"%d", &aux);  
			/*fscanf(fichero,"%d", &pmat[k]);*/
			/* printf("%d\t",pmat[k]); */
			//printf("%d\t",aux); 
			pmat[i][j]=aux;

			

		}
	printf("\n");
	fscanf(fichero, "\n"); 
	
	}
	

       
   fclose(fichero);
}

//-------------Funcion sumar velocidad

__global__ void sumarvelocidad(float ** pdist,int ** pvec,float ** psum, int node) {
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion
int k;
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
// int offset = x + y * blockDim.x * gridDim.x;

if (x<node){ //para que se paralelice en cada nodo
	if (y<nvec){	//para que se paralelice en cada vecino
		for(k=0;k<ndist;k++){	//para cada velocidad realizo la suma al no saber como paralelizar esta parte
			psum[x][k]+=pdist[pvec[x][y]][k];				
						}
		}


		}

}







//matriz de vecinos
void liberaVecinos(int ** pmat, int node) {
	int i;
	for( i=0; i<node; i++)
		free(pmat[i]);
		free(pmat);
	return;
	}

int ** allocaVecinos(int node, int nvecinos) {	
	int i;
	int **pmat;
	pmat = (int **) malloc(node * sizeof(int*));
	if( pmat == NULL )
	return NULL; 

	for( i=0; i<node; i++){
		pmat[i]=(int *) malloc(nvecinos * sizeof(int));
			if( pmat[i] == NULL ){
				liberaVecinos(pmat,i);
	return NULL;
	}
	}
	return pmat;
}


//matriz distribicion y matriz de suma
void liberaMatriz(float** pmat, int node) {
	int i;
	for( i=0; i<node; i++)
		free(pmat[i]);
		free(pmat);
	return;
	}
float ** allocaMatriz(int node, int nveloc) {	
	int i;
	float **pmat;
	pmat = (float **) malloc(node * sizeof(float*));
	if( pmat == NULL )
	return NULL; 

	for( i=0; i<node; i++){
		pmat[i]=(float *) malloc(nveloc * sizeof(float));
			if( pmat[i] == NULL ){
				liberaMatriz(pmat,i);
	return NULL;
	}
	}
	return pmat;
}




// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;
int nvel=9;

int *dev_vecinos;
float *dev_velocidad;
float *dev_suma;



// reserva en el host

int** hst_vecinos   = allocaVecinos(node,nvec);
float** hst_velocidad = allocaMatriz(node,nvel);
float** hst_suma      = allocaMatriz(node,nvel);


// reserva en el device
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int));
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(float));
hipMalloc( (void**)&dev_suma, node*nvel*sizeof(float));


// inicializacion de datos
printf("Llamo a llenarVecinos \n");

llenarVecinos(hst_vecinos, row, colum);
llenarVelocidad(hst_velocidad, row, colum);



// ver que es lo que tienen las matrices que se acaban de llenar en la alocacion
int i=0;
int j=0;
for(i=0;i<node;i++){
		for(j=0;j<nvec;j++){
			printf("%d\t",&hst_vecinos[i][j]); 
		}
		printf("\n");
}












// declaracion de eventos
hipEvent_t start;
hipEvent_t stop;
// creacion de eventos
hipEventCreate(&start);
hipEventCreate(&stop);
// marca de inicio
hipEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------

	printf("Llamo al Kernel \n");

	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);
   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------


// marca de final
hipEventRecord(stop,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
hipEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
hipEventDestroy(start);
hipEventDestroy(stop);






// copia de datos
hipMemcpy(hst_velocidad, dev_velocidad, node*nvel*sizeof(float), hipMemcpyDeviceToHost);
// salida


hipFree( dev_vecinos );
hipFree( dev_velocidad );
hipFree( hst_velocidad );

liberaMatriz(hst_suma,node);
liberaMatriz(hst_velocidad,node);
liberaVecinos(hst_vecinos,node);

return 0;
}
