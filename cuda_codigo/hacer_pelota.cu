// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar "velocidad"

void llenarVelocidad(float ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvel=9;
	float leer;
	
 	fichero = fopen("matriz_con_func_dist.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvel;j++){

			fscanf(fichero,"%f",&leer);	
			pmat[i][j]=leer;
			//printf("%f",leer);
		}
	fscanf(fichero, "\n"); 
	}
	
	fclose(fichero);

//--------------------------------------------------------------------------------------------------------------------------------------


    

//---------------------------------------------------- COMPROBACION DE QUE ESTA LEYENDO CORRECTAMENTE-----------------------------------
  /*  printf( "Contenido del fichero:\n" );
   for (i = 0; i < node; i++) {
      for (j = 0; j < nvel; j++)
	 printf ("%f ", pmat[i][j]);
      printf ("\n");
   }*/

}      


//-------------Funcion llenar los vecinos

void llenarVecinos(int ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvec=9;
		

	
 	fichero = fopen("matriz_con_vecinos.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvec;j++){

			fscanf(fichero,"%i",&pmat[i][j]);	
		}
	fscanf(fichero, "\n"); 
	}


       
   fclose(fichero);

//---------------------------------------------------- COMPROBACION DE QUE ESTA LEYENDO CORRECTAMENTE-----------------------------------

/*   printf( "Contenido del fichero:\n" );
   for (i = 0; i < node; i++) {
      for (j = 0; j < nvec; j++)
	 printf ("%d ", pmat[i][j]);
      printf ("\n");
   }*/

   return ;
}


//matriz de vecinos
void liberaVecinos(int ** pmat, int node) {
	int i;
	for( i=0; i<node; i++)
		free(pmat[i]);
		free(pmat);
	return;
	}

int ** allocaVecinos(int node, int nvecinos) {	
	int i;
	int **pmat;
	pmat = (int **) malloc(node * sizeof(int*));
	if( pmat == NULL )
	return NULL; 

	for( i=0; i<node; i++){
		pmat[i]=(int *) malloc(nvecinos * sizeof(int));
			if( pmat[i] == NULL ){
				liberaVecinos(pmat,i);
	return NULL;
	}
	}
	return pmat;
}


//matriz distribicion y matriz de suma
void liberaMatriz(float** pmat, int node) {
	int i;
	for( i=0; i<node; i++)
		free(pmat[i]);
		free(pmat);
	return;
	}
float ** allocaMatriz(int node, int nveloc) {	
	int i;
	float **pmat;
	pmat = (float **) malloc(node * sizeof(float*));
	if( pmat == NULL )
	return NULL; 

	for( i=0; i<node; i++){
		pmat[i]=(float *) malloc(nveloc * sizeof(float));
			if( pmat[i] == NULL ){
				liberaMatriz(pmat,i);
	return NULL;
	}
	}
	return pmat;
}



//-------------Funcion sumar velocidad

__global__ void sumarvelocidad(float ** pdist,int ** pvec,float ** psum, int node) {
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion
int k=0;
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
printf("Estoy en el kernel \n\n\n\n");
if (x<node){ //para que se paralelice en cada nodo
	if (y<nvec){	//para que se paralelice en cada vecino
		for(k=0;k<ndist;k++){	//para cada velocidad realizo la suma al no saber como paralelizar esta parte
			psum[x][k]+=pdist[pvec[x][y]][k];				
						}
		}


		}
printf("Termine el kernel \n\n\n\n");
}











// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;
int nvel=9;

int *dev_vecinos;
float *dev_velocidad;
float *dev_suma;



// reserva en el host

int** hst_vecinos   = allocaVecinos(node,nvec);
float** hst_velocidad = allocaMatriz(node,nvel);
float** hst_suma      = allocaMatriz(node,nvel);


// reserva en el device
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int));
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(float));
hipMalloc( (void**)&dev_suma, node*nvel*sizeof(float));


// inicializacion de datos
llenarVecinos(hst_vecinos, row, colum);
llenarVelocidad(hst_velocidad, row, colum);

// pasaje de los datos del hst al dev 

hipMemcpy(dev_vecinos,hst_vecinos,node*nvel*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_velocidad,hst_velocidad,node*nvel*sizeof(float),hipMemcpyHostToDevice);


printf("Termine de pasar los datos al dev \n\n\n\n");


/*
// ver que es lo que tienen las matrices que se acaban de llenar en la alocacion
int i=0;
int j=0;
for(i=0;i<node;i++){
		for(j=0;j<nvec;j++){
			printf("%d\t",&hst_vecinos[i][j]); 
		}
		printf("\n");
}


printf("Que hay en el dev \n\n\n\n");

*/







// declaracion de eventos
hipEvent_t start;
hipEvent_t stop;
// creacion de eventos
hipEventCreate(&start);
hipEventCreate(&stop);
// marca de inicio
hipEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------

	printf("Llamo al Kernel \n");

//	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);
	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);
   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------


// ver si esta sumando bien
/*   printf( "Contenido de SUMA :\n" );

int i=0;
int j=0;

   for (i = 0; i < node; i++) {
      for (j = 0; j < nvec; j++)
	 printf ("%d ", hst_suma[i][j]);
      printf ("\n");
   }*/









// marca de final
hipEventRecord(stop,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
hipEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
hipEventDestroy(start);
hipEventDestroy(stop);






// copia de datos
hipMemcpy(hst_velocidad, dev_velocidad, node*nvel*sizeof(float), hipMemcpyDeviceToHost);
// salida


hipFree( dev_vecinos );
hipFree( dev_velocidad );
hipFree( hst_velocidad );

liberaMatriz(hst_suma,node);
liberaMatriz(hst_velocidad,node);
liberaVecinos(hst_vecinos,node);

return 0;
}
