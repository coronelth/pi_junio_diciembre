// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


//-------------Funcion sumar velocidad

__global__ void densidad_suma_doble_if(float * pdist,float * psum, int node) {

int ndist=9;	//numero de funcion de distribucion
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int i;

if (x<node){ //para que se paralelice en cada nodo
	/*if (y<ndist){	//para que se paralelice en cada parte del vector de funcion de distribucion
		psum[x]+=  pdist[(x*ndist+y)];} */
		for(i=0;i<ndist;i++)	
			psum[x]+=  pdist[(x*ndist+i)];
		}
}

// nodo == x
//velocidad == y
