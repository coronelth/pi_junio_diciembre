// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


//-------------Funcion sumar velocidad

__global__ void densidad_suma_doble_if(float * pdist,float * psum, int node) {

int ndist=9;	//numero de funcion de distribucion
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x<node){ //para que se paralelice en cada nodo
	if (y<ndist){	//para que se paralelice en cada parte del vector de funcion de distribucion
		psum[x]+=  pdist[(x*ndist+y)];} 
		}
}

// nodo == x
//velocidad == y
