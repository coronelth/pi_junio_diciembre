#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "llenar_vecinos.h"


int main(int argc, char** argv) {


    int row=6;
    int colum=6;
    int node=row*colum;
    int nvec=9;
    int nvel=9;

    int *dev_vecinos;
    double *dev_velocidad;
    double *dev_suma;


    double *hst_velocidad;
    int *hst_vecinos;


    // reserva en el host
    hst_velocidad = (double*)malloc( node*nvel*sizeof(double) );
    hst_vecinos   = (int*)malloc( node*nvec*sizeof(int) );

    // reserva en el device
    hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(double) );
    hipMalloc( (void**)&dev_suma, node*nvel*sizeof(double) );
    hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int) );

    // inicializacion de datos
    llenarVecinos(&hst_vecinos, row, colum);



    // Liberacion de memoria

    hipFree( dev_vecinos );
    hipFree( dev_velocidad );
    hipFree( hst_velocidad );

    free(hst_velocidad);
    free(hst_vecinos);


    return 0;
}
