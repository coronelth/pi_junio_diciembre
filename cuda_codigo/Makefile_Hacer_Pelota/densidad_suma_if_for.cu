// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


//-------------Funcion sumar velocidad

__global__ void densidad_suma_if_for(float * pdist,float * psum, int node) {

int ndist=9;	//numero de funcion de distribucion
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int i;

if (x<node){ //para que se paralelice en cada nodo
		for(i=0;i<ndist;i++)	
			psum[x]+=  pdist[(x*ndist+i)];
		}
}

// nodo == x
//velocidad == y
