// includes


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//-------------Funcion llenar los vecinos

void llenarVecinos(int * pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvec=9;
		

	
 	fichero = fopen("matriz_con_vecinos.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvec;j++){

			fscanf(fichero,"%i",&pmat[i*nvec+j]);	
		}
	fscanf(fichero, "\n"); 
	}


       
   fclose(fichero);

//---------------------------------------------------- COMPROBACION DE QUE ESTA LEYENDO CORRECTAMENTE-----------------------------------

/*   printf( "Contenido de :\n" );
   for (i = 0; i < node; i++) {
      for (j = 0; j < nvec; j++)
	 printf ("%d ", pmat[i*nvec+j]);
      printf ("\n");
   }*/

   return ;
}
