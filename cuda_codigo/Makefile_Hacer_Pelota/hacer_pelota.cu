// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "llenar_vecinos.h"
#include "alloca_vecinos.h"
#include "libera_vecinos.h"
#include "alloca_vector.h"
#include "libera_vector.h"
#include "llenar_velocidad.h"
#include "sumar_velocidad.h"
#include "guardar_suma.h"
#include "densidad_suma_if_for.h"
#include "densidad_suma_doble_if.h"
#include "llenar_suma.h"

#include "generator_mat_vecinos.h"
#include "generator_mat_dist.h"

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row   = 6;
int colum = 6;
int node  = row*colum;
int nvec  = 9;
int nvel  = 9;
int one   = 1;
int i;

int *dev_vecinos;
float *dev_velocidad;
float *dev_suma;

// Generacion de la matriz de vecinos y pdist

generator_matriz_dist (row, colum);
generator_matriz_vecinos (row, colum);

// reserva en el host

int* hst_vecinos   = allocaVecinos(node,nvec);
float* hst_velocidad = allocaVector(node,nvel);
float* hst_suma      = allocaVector(node,one);

// reserva en el device
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int));
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(float));
hipMalloc( (void**)&dev_suma, node*sizeof(float));


// inicializacion de datos
llenarVecinos(hst_vecinos, row, colum);
llenarVelocidad(hst_velocidad, row, colum);

llenarSuma(hst_suma,node);
/*
for(i=0;i<node;i++){ // inicializar en cero el valor de la hst_suma
	hst_suma[i] = 0;
}
*/

// pasaje de los datos del hst al dev 

hipMemcpy(dev_vecinos,hst_vecinos,node*nvel*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_velocidad,hst_velocidad,node*nvel*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(dev_suma,hst_suma,node*one*sizeof(float),hipMemcpyHostToDevice);

/*
// ------------------- ver que se este realizando bien el pasaje de los datos

liberaVecinos(hst_vecinos);
liberaVector(hst_suma);
liberaVector(hst_velocidad);

int*   hst_vecinos_p   = allocaVecinos(node,nvec);
float* hst_velocidad_p = allocaVector(node,nvel);
float* hst_suma_p      = allocaVector(node,nvel);

hipMemcpy(hst_vecinos_p, dev_vecinos, node*nvel*sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(hst_velocidad_p, dev_velocidad, node*nvel*sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(hst_suma_p, dev_suma, node*nvel*sizeof(int), hipMemcpyDeviceToHost);

// ver que es lo que tienen las matrices que se acaban de llenar en la alocacion

int j=0;
for(i=0;i<node;i++){
		for(j=0;j<nvec;j++){
			printf("%d\t",hst_vecinos_p[i*nvel+j]); 
		}
		printf("\n");
}
printf("\n\n");


for(i=0;i<node;i++){
		for(j=0;j<nvec;j++){
			printf("%f\t",hst_velocidad_p[i*nvel+j]); 
		}
		printf("\n");
}
printf("\n\n");

for(i=0;i<node;i++){
		for(j=0;j<nvec;j++){
			printf("%f\t",hst_suma_p[i*nvel+j]); 
		}
		printf("\n");
}
printf("\n\n");
*/


// declaracion de eventos
hipEvent_t start;
hipEvent_t stop;
// creacion de eventos
hipEventCreate(&start);
hipEventCreate(&stop);
// marca de inicio
hipEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------

	densidad_suma_if_for<<<64,256>>>(dev_velocidad,dev_suma,node); 
   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------

// marca de final
hipEventRecord(stop,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
hipEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
hipEventDestroy(start);
hipEventDestroy(stop);

// copia de datos
hipMemcpy(hst_suma, dev_suma, node*one*sizeof(float), hipMemcpyDeviceToHost);
// salida

// ver si esta sumando bien
   printf( "Contenido de SUMA :\n" );


for(i=0;i<node;i++){
		printf("%f\t",hst_suma[i]); 
		printf("\n");		
		
}
printf("\n\n");
//----------------------------------------------------------Realizo lo mismo para el otro kernel propuesto-----------------------------------------------------------------------

// declaracion de eventos
hipEvent_t start_p;
hipEvent_t stop_p;
// creacion de eventos
hipEventCreate(&start_p);
hipEventCreate(&stop_p);
// marca de inicio
hipEventRecord(start_p,0);
// codigo a temporizar en el device

// -----------------------------------------------

	densidad_suma_doble_if<<<64,256>>>(dev_velocidad,dev_suma,node); 
   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------

// marca de final
hipEventRecord(stop_p,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop_p);
// calculo del tiempo en milisegundos
float elapsedTime_p;
hipEventElapsedTime(&elapsedTime_p,start_p,stop_p);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime_p);
// liberacion de recursos
hipEventDestroy(start_p);
hipEventDestroy(stop_p);

// copia de datos
hipMemcpy(hst_suma, dev_suma, node*one*sizeof(float), hipMemcpyDeviceToHost);
// salida


// ver si esta sumando bien
   printf( "Contenido de SUMA :\n" );


for(i=0;i<node;i++){
		printf("%f\t",hst_suma[i]); 
		printf("\n");		
		
}
printf("\n\n");



//----------------------------------------------------------------------------------------------------------------------------------------------------------------------
guardar_suma(hst_suma,node,one);


hipFree( dev_vecinos );
hipFree( dev_velocidad );
hipFree( hst_velocidad );


liberaVector(hst_suma);
liberaVector(hst_velocidad);
liberaVecinos(hst_vecinos);





return 0;
}
