// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "llenar_vecinos.h"
#include "alloca_vecinos.h"
#include "libera_vecinos.h"
#include "alloca_vector.h"
#include "libera_vector.h"
#include "llenar_velocidad.h"
#include "sumar_velocidad.h"

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;
int nvel=9;

int *dev_vecinos;
float *dev_velocidad;
float *dev_suma;


/*
// reserva en el host

int** hst_vecinos   = allocaVecinos(node,nvec);
float** hst_velocidad = allocaMatriz(node,nvel);
float** hst_suma      = allocaMatriz(node,nvel);
*/




// reserva en el host

int*   hst_vecinos   = allocaVecinos(node,nvec);
float* hst_velocidad = allocaVector(node,nvel);
float* hst_suma      = allocaVector(node,nvel);



// reserva en el device
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int));
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(float));
hipMalloc( (void**)&dev_suma, node*nvel*sizeof(float));


// inicializacion de datos
llenarVecinos(hst_vecinos, row, colum);
llenarVelocidad(hst_velocidad, row, colum);

// pasaje de los datos del hst al dev 

hipMemcpy(dev_vecinos,hst_vecinos,node*nvel*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_velocidad,hst_velocidad,node*nvel*sizeof(float),hipMemcpyHostToDevice);


printf("Termine de pasar los datos al dev \n\n\n\n");


/*
// ver que es lo que tienen las matrices que se acaban de llenar en la alocacion
int i=0;
int j=0;
for(i=0;i<node;i++){
		for(j=0;j<nvec;j++){
			printf("%d\t",&hst_vecinos[i][j]); 
		}
		printf("\n");
}


printf("Que hay en el dev \n\n\n\n");

*/







// declaracion de eventos
hipEvent_t start;
hipEvent_t stop;
// creacion de eventos
hipEventCreate(&start);
hipEventCreate(&stop);
// marca de inicio
hipEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------

	printf("Llamo al Kernel \n");

//	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);
	sumarvelocidad<<<16,256>>>(dev_velocidad, dev_vecinos, dev_suma, node);
   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------


// ver si esta sumando bien
/*   printf( "Contenido de SUMA :\n" );

int i=0;
int j=0;

   for (i = 0; i < node; i++) {
      for (j = 0; j < nvec; j++)
	 printf ("%d ", hst_suma[i][j]);
      printf ("\n");
   }*/

// marca de final
hipEventRecord(stop,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
hipEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
hipEventDestroy(start);
hipEventDestroy(stop);






// copia de datos
hipMemcpy(hst_velocidad, dev_velocidad, node*nvel*sizeof(float), hipMemcpyDeviceToHost);
// salida


hipFree( dev_vecinos );
hipFree( dev_velocidad );
hipFree( hst_velocidad );

/*
liberaMatriz(hst_suma,node);
liberaMatriz(hst_velocidad,node);
liberaVecinos(hst_vecinos,node);
*/



liberaVector(hst_suma);
liberaVector(hst_velocidad);
liberaVecinos(hst_vecinos);














return 0;
}
