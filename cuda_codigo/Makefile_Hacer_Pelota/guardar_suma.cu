
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

void guardar_suma(float *pmat,int node, int ndist ){

// Guardar el archivo en un .txt
//-----------------------------------------------
int i;
int j;

FILE *f = fopen("suma_de_coliciones.txt", "w");
if (f == NULL)
{
    printf("Error opening file!\n");
    exit(1);
}

for(i=0;i<node;i++){
	for(j=0;j<ndist;j++){
		fprintf(f,"%f\t",pmat[i*ndist+j] );
		}
		
	fprintf(f,"\n");
	
}

fclose(f);

	return ;
}
