// includes


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// Funciones de alocacion  para el vector que representa la Matriz

float *allocaVector(int node, int n) {
	float *v = (float *) malloc(node * n * sizeof(float));
	assert(v != NULL);
	return v;
}
