// includes


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//-------------Funcion llenar "velocidad"

void llenarVelocidad(float * pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvel=9;
	float leer;
	
 	fichero = fopen("matriz_con_func_dist.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvel;j++){

			fscanf(fichero,"%f",&leer);	
			pmat[ i*nvel + j]=leer;
			//printf("%f",leer);
		}
	fscanf(fichero, "\n"); 
	}
	
	fclose(fichero);

//--------------------------------------------------------------------------------------------------------------------------------------


    

//---------------------------------------------------- COMPROBACION DE QUE ESTA LEYENDO CORRECTAMENTE-----------------------------------
  /*  printf( "Contenido del fichero:\n" );
   for (i = 0; i < node; i++) {
      for (j = 0; j < nvel; j++)
	 printf ("%f ", pmat[ i*nvel + j]);
      printf ("\n");
   }*/

}      
