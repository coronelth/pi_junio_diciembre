#include "hip/hip_runtime.h"
// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


//-------------Funcion sumar velocidad

__global__ void sumarvelocidad(float * pdist,int ** pvec,float ** psum, int node) {

printf("Estoy en el kernel \n\n\n\n");
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion
int k=0;
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x<node){ //para que se paralelice en cada nodo
	if (y<nvec){	//para que se paralelice en cada vecino
		for(k=0;k<ndist;k++){	//para cada velocidad realizo la suma al no saber como paralelizar esta parte
			psum[x][k]+=pdist[pvec[x][y]][k];				
						}
		}


		}
printf("Termine el kernel \n\n\n\n");
}

