// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


//-------------Funcion sumar velocidad

__global__ void sumarvelocidad(float * pdist,int * pvec,float * psum, int node) {

int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion
int k=0;
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x<node){ //para que se paralelice en cada nodo
	if (y<nvec){	//para que se paralelice en cada vecino
		for(k=0;k<ndist;k++){	//para cada velocidad realizo la suma al no saber como paralelizar esta parte
			psum[(x*ndist+k)]+=  pdist[((pvec[(x*nvec+y)])*ndist+k)];
				
						}
		}


		}
}

// nodo == x
//vecino == y
//velocidad == k
