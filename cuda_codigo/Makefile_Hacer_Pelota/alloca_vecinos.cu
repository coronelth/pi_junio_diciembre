// includes


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

int * allocaVecinos(int node, int nvecinos){
	int *v = (int *) malloc(node * nvecinos * sizeof(int));
	assert(v != NULL);
	return v;
}
