// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar "velocidad"
void llenarVelocidad(double ** pmat, int row, int colum);
//-------------Funcion llenar los vecinos
void llenarVecinos(int ** pmat, int row, int colum);
//-------------Funcion sumar velocidad
__global__ void sumarvelocidad(double ** pdist,int ** pvec,double ** psum, int node); 



// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;
int nvel=9;

int *dev_vecinos;
int *hst_vec;
double *dev_velocidad;
double *dev_suma;
double *hst_velocidad;


// reserva en el host
hst_velocidad = (double*)malloc( node*nvel*sizeof(double) );

// esto se implemento ---> hst_vec
hst_vec = (int*)malloc( node*nvel*sizeof(int) );


// reserva en el device
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(double) );
hipMalloc( (void**)&dev_suma, node*nvel*sizeof(double) );
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int) );
printf("\n\nvoy bien\n\n");




// inicializacion de datos
printf("ccccc");

llenarVecinos(&hst_vec, row, colum);
//llenarVecinos(&dev_vecinos, row, colum);

hipMemcpy(dev_velocidad, hst_vec , node*nvec*sizeof(int), hipMemcpyHostToDevice);

printf("ccccc");


/*

llenarVelocidad(&dev_velocidad, row, colum);


// declaracion de eventos
cudaEvent_t start;
cudaEvent_t stop;
// creacion de eventos
cudaEventCreate(&start);
cudaEventCreate(&stop);
// marca de inicio
cudaEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------
	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);



   if (cudaDeviceSynchronize() != cudaSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------


// marca de final
cudaEventRecord(stop,0);
// sincronizacion GPU-CPU
cudaEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
cudaEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
cudaEventDestroy(start);
cudaEventDestroy(stop);






// copia de datos
cudaMemcpy(hst_velocidad, dev_velocidad, node*nvel*sizeof(float), cudaMemcpyDeviceToHost);
// salida


cudaFree( dev_vecinos );
cudaFree( dev_velocidad );
cudaFree( hst_velocidad );*/


return 0;
}








