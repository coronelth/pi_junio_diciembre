// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar "velocidad"
void llenarVelocidad(double ** pmat, int row, int colum);
//-------------Funcion llenar los vecinos
void llenarVecinos(int ** pmat, int row, int colum);
//-------------Funcion sumar velocidad
__global__ void sumarvelocidad(double ** pdist,int ** pvec,double ** psum, int node); 



// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;
int nvel=9;

int *dev_vecinos;
double *dev_velocidad;
double *dev_suma;
double *hst_velocidad;


// reserva en el host
hst_velocidad = (double*)malloc( node*nvel*sizeof(double) );

// reserva en el device
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(double) );
hipMalloc( (void**)&dev_suma, node*nvel*sizeof(double) );
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int) );

// inicializacion de datos

llenarVecinos(&dev_vecinos, row, colum);
llenarVelocidad(&dev_velocidad, row, colum);


// declaracion de eventos
hipEvent_t start;
hipEvent_t stop;
// creacion de eventos
hipEventCreate(&start);
hipEventCreate(&stop);
// marca de inicio
hipEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------
	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);



   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------


// marca de final
hipEventRecord(stop,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
hipEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
hipEventDestroy(start);
hipEventDestroy(stop);






// copia de datos
hipMemcpy(hst_velocidad, dev_velocidad, node*nvel*sizeof(float), hipMemcpyDeviceToHost);
// salida


hipFree( dev_vecinos );
hipFree( dev_velocidad );
hipFree( hst_velocidad );


return 0;
}
