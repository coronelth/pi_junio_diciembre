// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar la "velocidad"





//-------------Funcion llenar los vecinos



//-------------Funcion sumar velocidad

__device__ void sumarvelocidad(double ** pdist,int ** pvec,double ** psum, int node) {
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

int j;

if (x<node){
	if (y<nvec){
		j=y;//para que una parte haga los vecinos y la otra parte calcule la suma de los 
		psum[x][k]+=pdist[pvec[x][j]][k];
		if ((y > nvec)&&(y < nvec+ndist )){
			psum[x][k]+=pdist[pvec[x][j]][k];			
			}
		}
}

}







// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;

int *dev_vecinos;
float *dev_matriz;
float *hst_matriz;


// reserva en el host
hst_matriz = (float*)malloc( node*nvec*sizeof(float) );

// reserva en el device
hipMalloc( (void**)&dev_matriz, node*nvec*sizeof(float) );
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int) );

// inicializacion de datos


/*
// copia de datos
hipMemcpy(dev_matriz, hst_matriz, N*N*sizeof(float), hipMemcpyHostToDevice);
// salida
*/


hipFree( dev_vecinos )
hipFree( dev_matriz )
hipFree( hst_matriz )


return 0;
}
