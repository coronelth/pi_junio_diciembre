// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar la "velocidad"

void llenarVelocidad(int ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvel=9;
		

	
 	fichero = fopen("matriz_con_func_dist.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvel;j++){

			fscanf(fichero,"%i",&pmat[i][j]);	
		}
	fscanf(fichero, "\n"); 
	}

       
   fclose(fichero);


//-------------Funcion llenar los vecinos

void llenarVecinos(int ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvec=9;
		

	
 	fichero = fopen("matriz_con_vecinos.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvec;j++){

			fscanf(fichero,"%i",&pmat[i][j]);	
		}
	fscanf(fichero, "\n"); 
	}


       
   fclose(fichero);


//-------------Funcion sumar velocidad

__device__ void sumarvelocidad(double ** pdist,int ** pvec,double ** psum, int node) {
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

int j;

if (x<node){
	if (y<nvec){
		j=y;//para que una parte haga los vecinos y la otra parte calcule la suma de los 
		psum[x][k]+=pdist[pvec[x][j]][k];
		if ((y > nvec)&&(y < nvec+ndist )){
			psum[x][k]+=pdist[pvec[x][j]][k];			
			}
		}
}

}







// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;

int *dev_vecinos;
float *dev_matriz;
float *hst_matriz;


// reserva en el host
hst_matriz = (float*)malloc( node*nvec*sizeof(float) );

// reserva en el device
hipMalloc( (void**)&dev_matriz, node*nvec*sizeof(float) );
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int) );

// inicializacion de datos


/*
// copia de datos
hipMemcpy(dev_matriz, hst_matriz, N*N*sizeof(float), hipMemcpyHostToDevice);
// salida
*/


hipFree( dev_vecinos )
hipFree( dev_matriz )
hipFree( hst_matriz )


return 0;
}
