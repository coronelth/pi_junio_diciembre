// includes

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//-------------Funcion llenar "velocidad"

void llenarVelocidad(float ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvel=9;
	float leer;

	//float **pmat = NULL;	

 	fichero = fopen("matriz_con_func_dist.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }


	printf("Estoy por leer del archivo matriz_con_func_dist.txt \n");		
	
	for(i=0;i<node;i++){
		
		printf("Estoy por entrar al for con i...");

		for(j=0;j<nvel;j++){

			fscanf(fichero,"%f",&leer);
			printf("Imprimo leer= %f  y cargo pmat[i][j]");	
			pmat[i][j]=leer;
			printf("%f",leer);
		}
	fscanf(fichero, "\n"); 
	}

	fclose(fichero);

	for(i=0;i<node;i++){
		for(j=0;j<nvel;j++){
			printf("%f\t",pmat[i][j]);
		}
	}
}


//-------------Funcion llenar los vecinos

void llenarVecinos(int ** pmat, int row, int colum){
	FILE *fichero;
	int node=row*colum;
	int i,j;
	int nvec=9;
	int aux=0;

	int k=0;

	/*pmat[k]=0;*/

 	fichero = fopen("matriz_con_vecinos.txt","r");
	
	   if (fichero==NULL)
   	{
  	    printf( "No se puede abrir el fichero.\n" );
	      system("pause");
	      exit (EXIT_FAILURE);
	   }
	
	for(i=0;i<node;i++){

		for(j=0;j<nvec;j++){

			fscanf(fichero,"%d", &aux);  
			/*fscanf(fichero,"%d", &pmat[k]);*/
			/* printf("%d\t",pmat[k]); */
			printf("%d\t",aux); 
			/*pmat[k]=aux;*/

			k=k+1;

		}
/*	printf("\n");*/
	fscanf(fichero, "\n"); 
	
	}
	

       
   fclose(fichero);
}

//-------------Funcion sumar velocidad

__global__ void sumarvelocidad(float ** pdist,int ** pvec,float ** psum, int node) {
int nvec=9;	//numero de vecinos
int ndist=9;	//numero de funcion de distribucion
int k;
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
// int offset = x + y * blockDim.x * gridDim.x;

if (x<node){ //para que se paralelice en cada nodo
	if (y<nvec){	//para que se paralelice en cada vecino
		for(k=0;k<ndist;k++){	//para cada velocidad realizo la suma al no saber como paralelizar esta parte
			psum[x][k]+=pdist[pvec[x][y]][k];				
						}
		}


		}

}

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

// declaracion
int row=6;
int colum=6;
int node=row*colum;
int nvec=9;
int nvel=9;

int *dev_vecinos;
float *dev_velocidad;
float *dev_suma;
float *hst_velocidad;


// reserva en el host
hst_velocidad = (float*)malloc( node*nvel*sizeof(float) );

// reserva en el device
hipMalloc( (void**)&dev_velocidad, node*nvel*sizeof(float) );
hipMalloc( (void**)&dev_suma, node*nvel*sizeof(float) );
hipMalloc( (void**)&dev_vecinos, node*nvec*sizeof(int) );

// inicializacion de datos
printf("Llamo a llenarVecinos \n");
llenarVecinos(&dev_vecinos, row, colum);

llenarVelocidad(&dev_velocidad, row, colum);


// declaracion de eventos
hipEvent_t start;
hipEvent_t stop;
// creacion de eventos
hipEventCreate(&start);
hipEventCreate(&stop);
// marca de inicio
hipEventRecord(start,0);
// codigo a temporizar en el device

// -----------------------------------------------

	printf("Llamo al Kernel \n");

	sumarvelocidad<<<16,256>>>(&dev_velocidad, &dev_vecinos, &dev_suma, node);
   if (hipDeviceSynchronize() != hipSuccess) {
       fprintf (stderr, "Cuda call failed\n");
   }

// aqui va el kernel que realiza la suma que es lo que se quiere medir
// -----------------------------------------------


// marca de final
hipEventRecord(stop,0);
// sincronizacion GPU-CPU
hipEventSynchronize(stop);
// calculo del tiempo en milisegundos
float elapsedTime;
hipEventElapsedTime(&elapsedTime,start,stop);
// impresion de resultados
printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
// liberacion de recursos
hipEventDestroy(start);
hipEventDestroy(stop);






// copia de datos
hipMemcpy(hst_velocidad, dev_velocidad, node*nvel*sizeof(float), hipMemcpyDeviceToHost);
// salida


hipFree( dev_vecinos );
hipFree( dev_velocidad );
hipFree( hst_velocidad );


return 0;
}
